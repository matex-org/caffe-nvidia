#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  DLOG(INFO) << "FGPU Call";
#ifdef USE_DEEPMEM
  Batch<Dtype> * batch;
  volatile bool *dirtybit;
  PopBatch<Dtype>* p_batch;
  DLOG(INFO) << "FCPU Call DEEPMEM";
  if(cache_size_)
  {
// #ifndef CPU_ONLY
//     hipStream_t stream;
//     if (Caffe::mode() == Caffe::GPU) {
//       CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
//     }
// #endif
    // PopBatch<Dtype> p_batch;
    //Do we handle the refill on l1 cache?
    if(!caches_[0]->prefetch && caches_[0]->empty()) //empty cache
    {
      //LOG(INFO) << "Local Refill ";
      //Refill before poping using the policy we have
      (caches_[0]->*(caches_[0]->local_refill_policy))(1);
    }
    batch = l0cache_full_.pop("Prefetch cache queue empty (GPU)");
    dirtybit = dirtybit_.front();//("DirtyBit GPU");
    dirtybit_.pop();
  }
  else //Use the original unmofified code to get a batch
  {
    //int accuracySize = historical_accuracy.size();
    //for(int i=0; i< accuracySize; i++)
    //  LOG(INFO) << "ACC" << historical_accuracy[i];
    // Here for CPU we do transformation
    //if (Caffe::mode() == Caffe::CPU) {
    // if (!prefetch) {
    //   this->GetBatch();
    // }
    batch = prefetch_full_.pop("Prefetch cache queue empty (GPU)");
  }
#else
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
#endif

  // check batch has finished copying to the device
  CUDA_CHECK(hipStreamWaitEvent(hipStreamDefault, batch->copied_, 0));

  // Reshape to loaded data.
  if (this->transform_param_.use_gpu_transform()) {
    // instead of copy, perform out-of-place transform(!)
    this->data_transformer_->TransformGPU(top[0]->num(),
                                       top[0]->channels(),
                                       batch->data_.height(),
                                       batch->data_.width(),
                                       batch->data_.gpu_data(),
                                       top[0]->mutable_gpu_data(),
                                       batch->random_vec_.mutable_gpu_data());
  }  else {
    // Copy the data
    // Reshape to loaded data.
    top[0]->ReshapeLike(batch->data_);
    caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
               top[0]->mutable_gpu_data());
  }

  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
#ifdef USE_DEEPMEM
  if(cache_size_) {
  // We finished copy the batch so mark it for replacement
    *dirtybit = true;
    // *(p_batch->dirty) = true;
    // l0cache_free_.push(batch);
  }
  //Use the orginal code if caches are turned off
  if(cache_size_ == 0 || caches_[0]->size == 0)
    prefetch_free_.push(batch);
#else
  prefetch_free_.push(batch);
#endif
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
